#include "hip/hip_runtime.h"
/*
 * An exercise on the different types of memory available in CUDA
 */

#include <iostream>
#include <cstdlib>

// Error checking macro function
#define myCudaCheck(result) { cudaErrorCheck((result), __FILE__, __LINE__); }
inline void cudaErrorCheck(hipError_t err, const char* file, int line)
{
  if (err != hipSuccess) {
    std::cerr << "CUDA error: " << hipGetErrorString(err) << " at " << file << ":" << line << std::endl;
    exit(err);
  }
}

// Array size
// HANDSON 2.1 Change the array size to a static __constant__ int
#define ARRAY_SIZE 65536
static __constant__ int arr_size 65536;

// CUDA threads per block
#define nThreads 128

// Array reversing kernel
__global__
void reverse(float* devA, float* devB)
{
  // HANDSON 2.3 Create a __shared__ temporary array of length nThreads for the swap
  __shared__ float tmp_arr[nThreads];
  
  // Get the index in this block
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  // HANDSON 2.4 Fill the temporary array
  tmp_arr[nThreads - (threadIdx.x+1)] = devA[idx];

  // HANDSON 2.5 synchronize the threads
  __syncthreads();

  // HANDSON 2.6 Calculate the initial position of this block in the grid
  int blockOffset = arr_size - (blockIdx.x + 1) * blockDim.x;

  // Reverse the elements
  // HANDSON 2.7 Fill the output array with the reversed elements from this block
  devB[blockOffset + threadIdx.x] = tmp_arr[threadIdx.x];
}

// Main host function
int main( )
{
  // HANDSON 2.2 Replace the host array size by a const int
  //             Here and elsewhere
  // size of the array in char
  const int host_size = 65536;
  size_t sizeChar = host_size * sizeof(float);
  

  // Allocate host memory
  float* hostIn = (float*) malloc(sizeChar);
  float* hostOut = (float*) malloc(sizeChar);

  // Allocate device memory
  float* devIn;
  float* devOut;
  myCudaCheck(
	      hipMalloc(&devIn, sizeChar)
	      );
  myCudaCheck(
	      hipMalloc(&devOut, sizeChar)
	      );

  // Initialize the arrays
  for (int i = 0; i < ARRAY_SIZE; i++) {
    hostIn[i] = i;
    hostOut[i] = 0;
  }

  // Copy the input array from the host to the device
  myCudaCheck(
	      hipMemcpy(devIn, hostIn, sizeChar, hipMemcpyHostToDevice)
	      );

  // Define the size of the task
  dim3 blocksPerGrid(ARRAY_SIZE/nThreads);
  dim3 threadsPerBlock(nThreads);

  reverse<<<blocksPerGrid, threadsPerBlock>>>(devIn, devOut);

  // Wait for all threads to complete
  myCudaCheck(
	      hipDeviceSynchronize()
	      );

  // Copy the result array back to the host
  myCudaCheck(
	      hipMemcpy(hostOut, devOut, sizeChar, hipMemcpyDeviceToHost)
	      );

  // Check and print the result
  int nCorrect = 0;
  for (int i = 0; i < ARRAY_SIZE; i++) {
    nCorrect += (hostOut[i] == hostIn[ARRAY_SIZE - (i+1)]) ? 1 : 0;
  }
  std::cout << ((nCorrect == ARRAY_SIZE) ? "Success! " : "Failure: ");
  std::cout << nCorrect  << " elements were correctly swapped." << std::endl;

  // Free device memory
  myCudaCheck(
	      hipFree(devIn)
	      );
  myCudaCheck(
	      hipFree(devOut)
	      );

  // Free host memory
  free(hostIn);
  free(hostOut);

  return 0;
}