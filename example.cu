#include "hip/hip_runtime.h"
#include <iostream>
// __global__ specifier marks a GPU kernel. Returns void.
__global__

// saxpy stands for single precision a*x plus y.
// It is a combination of scalar multiplication and vector addition. 
void saxpy(int n, float a, float *x, float *y) {

  // threadIdx provides the location of the current thread in the block.
  // blockIdx provides the location of that block in the overall grid.
  // blockDim gives the number of threads in the block.
  // We can refer to a specific element in the grid by the expression below.
  int i = blockIdx.x*blockDim.x + threadIdx.x;

  // saxpy expression
  if (i < n) y[i] = a*x[i] + y[i];
}

int main(void) {

  // left shift operator
  // convert one to binary and then add 20 zeros to get 2^20
  int N = 1<<20;

  float *x, *y, *d_x, *d_y;

  // Alocate memory on the host.
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));

  // Allocates an array of size bytes on the device
  hipMalloc(&d_x, N*sizeof(float));
  hipMalloc(&d_y, N*sizeof(float));

  // fill x,y arrays with values.
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // hipMemcpy( destination array, source array, number of bytes to transfer, direction kind )
  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  // Perform SAXPY on 1M elements
  // The kernel parameters blocksPerGrid and threadsPerBlock define the sizeof the problem. 
  // syntax: kernelFunctionName<<<blocksPerGrid, threadsPerBlock>>>(arguments ...)
  saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

  // copy data from device to host.
  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  float maxError = 0.0f;

  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i]-4.0f)); // maxError expected to be 0.0 because y[i] is 4.0f

  std::cout << "Max error: " << maxError << std::endl;


  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);
}